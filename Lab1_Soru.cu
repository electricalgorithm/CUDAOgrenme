
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_addition(int *A,int *B,int *C,int size)//CUDA kernel
{
        // Thread ID'mizi alıyoruz.
        int id = (blockIdx.x * blockDim.x) + threadIdx.x;

        // Eğer Thread ID'si size'ı geçmiyorsa toplamayı yapıyoruz. 
        if (id < size)
                C[id] = A[id] + B[id];
}

int main()
{
        int size = 10000002;            //Dizi büyüklüğü (2.günki soru için 3'e tam bölünmeli)
        int ThreadPerBlock = 1024;      //Blok büyüklüğü. 32'nin katı olması iyi olur
        int BlockSize = (int) ceil((float)size/ThreadPerBlock); //@ Blok sayısı hesaplanacak
        printf("BlockSize = %d\n", BlockSize);

        int *A_Host,*B_Host,*C_Host;
        A_Host = new int[size];         //CPU belleğinde (Heap bölgesi) yer açılıyor
        B_Host = new int[size];         //CPU belleğinde (Heap bölgesi) yer açılıyor
        C_Host = new int[size];         //CPU belleğinde (Heap bölgesi) yer açılıyor

        for (int i = 1; i <= size; i++) //Diziye başlangıç değerleri atanıyor
        {
                A_Host[i-1] = i;
                B_Host[i-1] = 0;
        }

        int *A_GPU, *B_GPU, *C_GPU;
        //@ GPU Ana belleginde yer ayırılacak
        hipMalloc(&A_GPU, sizeof(int)*size);
        hipMalloc(&B_GPU, sizeof(int)*size);
        hipMalloc(&C_GPU, sizeof(int)*size);

        //@ Blok Büyüklüğü ve Grid Büyüklüğü dim3 türünde tanımlanacak
        dim3 DimBlock(ThreadPerBlock);
        dim3 DimGrid(BlockSize);

        hipEvent_t start, stop;        //Süre değişkenleri
        hipEventCreate(&start);
        hipEventCreate(&stop);
        float totaltime;                //Toplam süre değişkeni

        hipEventRecord(start); //Süre başlatıldı
	
	//@ CPU belleğinden GPU ana bellegine veri transferi gerçekleştirilecek
        hipMemcpy(A_GPU, A_Host, sizeof(int)*size, hipMemcpyHostToDevice);
        hipMemcpy(B_GPU, B_Host, sizeof(int)*size, hipMemcpyHostToDevice);

        //@ CUDA Kernel çalıştırılacak
        vector_addition<<<DimGrid, DimBlock>>>(A_GPU, B_GPU, C_GPU, size);

        //@ GPU ana belleğinden CPU bellegine veri transferi gerçekleştirilecek
        hipMemcpy(C_Host, C_GPU, sizeof(int)*size, hipMemcpyDeviceToHost);

        hipEventRecord(stop);          //Süre durduruldu
        hipEventSynchronize(stop);     //Event işlemleri bitene kadar program beklemekte
        hipEventElapsedTime(&totaltime, start, stop);  //Geçen süre hesaplanıyor
        printf("%f\n", totaltime);
        printf("%d\n", C_Host[size-1]);

        delete[] A_Host;        //Dizi CPU belleğinden siliniyor
        delete[] B_Host;        //Dizi CPU belleğinden siliniyor
        delete[] C_Host;        //Dizi CPU belleğinden siliniyor

        //@ Diziler GPU ana belleğinden silinecek
        hipFree(A_GPU);
        hipFree(B_GPU);
        hipFree(C_GPU);

        hipError_t err = hipGetLastError();//GPU'da oluşan son hatayı yakalıyor
        if ( err != hipSuccess )
                printf("CUDA Error: %s\n",hipGetErrorString(err));
}
